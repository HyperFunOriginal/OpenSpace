#include "hip/hip_runtime.h"
﻿#include "printstring_helper.h"
#include "gravitation.h"
#include "raymarch.h"

#include <chrono>
double one_second()
{
    std::chrono::steady_clock clock;
    const long long now = clock.now().time_since_epoch().count();
    _sleep(1000);
    return (double)(clock.now().time_since_epoch().count() - now);
}

// Implicit octree with morton indexing and a grid of sidelength 2^N, then raymarching straight through grids or collections thereof if there are no points there, i.e. cumsum over a certain patch is not increasing 
constexpr uint frames = 2100u;
constexpr float major_timestep = 40.f;
constexpr uint physics_substeps = 3u;

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    if (create_folder("SaveFolder"))
    {
        std::chrono::steady_clock clock;
        rng_state state(clock.now().time_since_epoch().count() >> 12u);
        gravitational_simulation simulation(3000000);
        smart_gpu_cpu_buffer<uint> temp(512 * 512);

        double step_second = physics_substeps * one_second();
        simulation.set_massive_sphere(3000000, 0, 5.97E+15f, 10000.f, make_float3(domain_size_km * .5f), make_float3(0.f), make_float3(0.f, 0.f, 7.2E-4f));
        
        for (uint i = 0u; i < frames; i++)
        {
            const long long now = clock.now().time_since_epoch().count();
            for (uint j = 0u; j < physics_substeps; j++)
            {
                simulation.sort_spatially();
                simulation.generate_gravitational_data();
                simulation.apply_gravitation();
                simulation.apply_kinematics(major_timestep / physics_substeps);
            }
            writeline("Saving image " + std::to_string(i) + ", Time taken per substep: " + std::to_string((clock.now().time_since_epoch().count() - now) * 1000.0 / step_second) + " ms");
            save_octree_image(temp, simulation, 512, 512, ("SaveFolder/" + std::to_string(i) + ".png").c_str());
        }
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
