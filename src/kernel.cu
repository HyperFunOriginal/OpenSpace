#include "hip/hip_runtime.h"
﻿#include "printstring_helper.h"
#include "cuda_headers/hydrodynamics_helper.h"
#include "cuda_headers/raymarch.h"

#include <chrono>
double one_second()
{
    std::chrono::steady_clock clock;
    const long long now = clock.now().time_since_epoch().count();
    _sleep(1000);
    return (double)(clock.now().time_since_epoch().count() - now);
}

// Tunable
constexpr float major_timestep = 10.f;
constexpr uint physics_substeps = 4u;
constexpr uint width  = 512u;
constexpr uint height = 512u;

uint obtain_particle_count()
{
    std::string line; writeline("Number of Particles?"); size_t particle_count = 100000u;
    std::getline(std::cin, line); particle_count = std::stoul(line);
    if (particle_count > 1000000u)
    {
        writeline("Are you sure you want to proceed with greater than 1 000 000 particles? \nRestart the program if not, and press enter otherwise.");
        std::getline(std::cin, line);
    }
    writeline("\nStarting Simulation:");
    return particle_count;
}

void run_grav_sim()
{
    if (create_folder("SaveFolder"))
    {
        std::chrono::steady_clock clock;
        rng_state state(clock.now().time_since_epoch().count() >> 12u);
        smart_gpu_cpu_buffer<uint> temp(width * height);

        uint particle_count = obtain_particle_count();
        double step_second = physics_substeps * one_second();
        gravitational_simulation simulation(particle_count);

        std::vector<initial_kinematic_object> v = std::vector<initial_kinematic_object>();
        v.push_back(initial_kinematic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 6378.f }, 6.0E+15f, domain_size_km * make_float3(.5f, .75f, .5f), make_float3(3.5f, 0.f, 0.f), make_float3(0.f, 0.f, 1.25e-3f)));
        v.push_back(initial_kinematic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 7000.f }, 7.9321666e+15f, domain_size_km * make_float3(.5f, .25f, .5f), make_float3(-3.5f, 0.f, 0.f), make_float3(0.f, 0.f, 1.25e-3f)));
        initialize_kinematic_objects(simulation, v);

        for (uint i = 0u; ; i++)
        {
            const long long now = clock.now().time_since_epoch().count();
            for (uint j = 0u; j < physics_substeps; j++)
            {
                simulation.sort_spatially();
                simulation.generate_gravitational_data();
                simulation.apply_gravitation();
                simulation.apply_kinematics_recenter(major_timestep / physics_substeps);
            }
            writeline("Saving image " + std::to_string(i) + ", Time taken per substep: " + std::to_string((clock.now().time_since_epoch().count() - now) * 1000.0 / step_second) + " ms");
            save_octree_image(temp, simulation, width, height, ("SaveFolder/" + std::to_string(i) + ".png").c_str());
        }
    }
}
void init_materials(hydrodynamics_simulation& simulation)
{
    // Iron
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].bulk_modulus_GPa = 170.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].limiting_heat_capacity_kJkgK = .47f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].standard_density_kgm3 = 7000.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].molar_mass_kgmol = 5.2E-2f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].thermal_scale_K = 300.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[0].stiffness_exponent = 4.6f;

    // Olivine
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].bulk_modulus_GPa = 120.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].limiting_heat_capacity_kJkgK = .7f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].standard_density_kgm3 = 4000.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].molar_mass_kgmol = 9E-2f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].thermal_scale_K = 100.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[1].stiffness_exponent = 4.6f;

    // Hydrogen
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].bulk_modulus_GPa = .193f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].limiting_heat_capacity_kJkgK = 10.14f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].standard_density_kgm3 = 86.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].molar_mass_kgmol = 2E-3f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].thermal_scale_K = 40.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[2].stiffness_exponent = 3.3f;

    // Water
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].bulk_modulus_GPa = 2.1f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].limiting_heat_capacity_kJkgK = 2.1f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].standard_density_kgm3 = 1000.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].molar_mass_kgmol = 1.8E-2f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].thermal_scale_K = 50.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[3].stiffness_exponent = 4.0f;

    // Helium
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].bulk_modulus_GPa = .03f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].limiting_heat_capacity_kJkgK = 3.2f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].standard_density_kgm3 = 210.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].molar_mass_kgmol = 4E-3f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].thermal_scale_K = 20.f;
    simulation.materials_cpu_copy.cpu_buffer_ptr[4].stiffness_exponent = 4.f;
    simulation.copy_materials_to_gpu();
}

void run_sph_sim()
{
    if (create_folder("SaveFolder"))
    {
        std::chrono::steady_clock clock;
        rng_state state(clock.now().time_since_epoch().count() >> 12u);
        smart_gpu_cpu_buffer<uint> temp(width * height);

        double step_second = physics_substeps * one_second();
        hydrogravitational_simulation simulation(1000000);
        smart_gpu_buffer<float3> x_factor(simulation.particle_capacity);
        init_materials(simulation);

        std::vector<initial_thermodynamic_object> v = std::vector<initial_thermodynamic_object>(); 
        v.push_back(initial_thermodynamic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 6400.f }, 6E+15f, domain_size_km * make_float3(.75f, .75f, .5f), make_float3(0.f), make_float3(0.f), 300.f, 1u));
        v.push_back(initial_thermodynamic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 5400.f }, 6E+15f, domain_size_km * make_float3(.25f, .25f, .5f), make_float3(0.f), make_float3(0.f), 300.f, 0u));
        v.push_back(initial_thermodynamic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 10000.f }, 8.3775804e+14f, domain_size_km * make_float3(.75f, .25f, .5f), make_float3(-1.f, 1.f, 0.f), make_float3(0.f), 300.f, 2u));
        v.push_back(initial_thermodynamic_object(initial_kinematic_object::geometry::GEOM_SPHERE, { 7000.f }, 2.8735101e+15f, domain_size_km * make_float3(.25f, .75f, .5f), make_float3(1.f, -1.f, 0.f), make_float3(0.f), 300.f, 3u));

        initialize_thermodynamic_objects(simulation, v);

        float average_time = 0.f;
        for (uint i = 0u; i < 6000; i++)
        {
            const long long now = clock.now().time_since_epoch().count();
            for (uint j = 0u; j < physics_substeps; j++)
                apply_xsph_variant(x_factor, simulation, major_timestep / physics_substeps, 5e-4f);
            double time = (clock.now().time_since_epoch().count() - now) * 1000.0 / step_second; average_time += time;
            writeline("Saving image " + std::to_string(i) + ", Time taken per substep: " + std::to_string(time) + " ms");
            save_octree_image(temp, simulation, width, height, ("SaveFolder/" + std::to_string(i) + ".png").c_str());
            if (i > 5u && time < average_time * .25f / i) // catch if something's gone off the rails
                break;
        }
    }
}

int main()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        Sleep(5000);
        return 1;
    }

    run_sph_sim();

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        Sleep(5000);
        return 1;
    }

    return 0;
}
